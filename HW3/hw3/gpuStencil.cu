#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>

#include "BC.h"

/**
 * Calculates the next finite difference step given a
 * grid point and step lengths.
 *
 * @param curr Pointer to the grid point that should be updated.
 * @param width Number of grid points in the x dimension.
 * @param xcfl Courant number for x dimension.
 * @param ycfl Courant number for y dimension.
 * @returns Grid value of next timestep.
 */
template<int order>
__device__
float Stencil(const float* curr, int width, float xcfl, float ycfl) {
    switch(order) {
        case 2:
            return curr[0] + xcfl * (curr[-1] + curr[1] - 2.f * curr[0]) +
                   ycfl * (curr[width] + curr[-width] - 2.f * curr[0]);

        case 4:
            return curr[0] + xcfl * (- curr[2] + 16.f * curr[1] - 30.f * curr[0] +
                                     16.f * curr[-1] - curr[-2]) + ycfl * (- curr[2 * width] +
                                             16.f * curr[width] - 30.f * curr[0] + 16.f * curr[-width] -
                                             curr[-2 * width]);

        case 8:
            return curr[0] + xcfl * (-9.f * curr[4] + 128.f * curr[3] -
                                     1008.f * curr[2] + 8064.f * curr[1] - 14350.f * curr[0] +
                                     8064.f * curr[-1] - 1008.f * curr[-2] + 128.f * curr[-3] -
                                     9.f * curr[-4]) + ycfl * (-9.f * curr[4 * width] +
                                             128.f * curr[3 * width] - 1008.f * curr[2 * width] +
                                             8064.f * curr[width] - 14350.f * curr[0] +
                                             8064.f * curr[-width] - 1008.f * curr[-2 * width] +
                                             128.f * curr[-3 * width] - 9.f * curr[-4 * width]);

        default:
            printf("ERROR: Order %d not supported", order);
            return HIP_NAN_F;
    }
}

/**
 * Kernel to propagate finite difference grid from the current
 * time point to the next.
 *
 * This kernel should be very simple and only use global memory.
 *
 * @param next[out] Next grid state.
 * @param curr Current grid state.
 * @param gx Number of grid points in the x dimension.
 * @param nx Number of grid points in the x dimension to which the full
 *           stencil can be applied (ie the number of points that are at least
 *           order/2 grid points away from the boundary).
 * @param ny Number of grid points in the y dimension to which th full
 *           stencil can be applied.
 * @param xcfl Courant number for x dimension.
 * @param ycfl Courant number for y dimension.
 */
template<int order>
__global__
void gpuStencil(float* next, const float* curr, int gx, int nx, int ny,
                float xcfl, float ycfl) {
    // assert((gx - nx) == order);
    // thread id inside (nx * ny) area
    int tid_x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int tid_y = (blockIdx.y * blockDim.y) + threadIdx.y;
    // thread id inside (gx * gy) area (pad with border)
    int gid_x = tid_x + order / 2;
    int gid_y = tid_y + order / 2;
    int index = gid_x + gx * gid_y;

    if (tid_x < nx && tid_y < ny)
        next[index] = Stencil<order>(curr + index, gx, xcfl, ycfl);
}

/**
 * Propagates the finite difference 2D heat diffusion solver
 * using the gpuStencil kernel.
 *
 * Use this function to do necessary setup and propagate params.iters()
 * number of times.
 *
 * @param curr_grid The current state of the grid.
 * @param params Parameters for the finite difference computation.
 * @returns Time required for computation.
 */
double gpuComputation(Grid& curr_grid, const simParams& params) {

    boundary_conditions BC(params);

    Grid next_grid(curr_grid);

    // declare variables and compute parameters
    const int nx = params.nx(), ny = params.ny();
    const double xcfl = params.xcfl(), ycfl = params.ycfl();
    const int gx = params.gx();
    const int order = params.order();

    // choose block size as 192 threads (organize them as square as possible)
    const unsigned int thread_num = 192;
    const unsigned int block_x = 32;
    const unsigned int block_y = thread_num / block_x;
    dim3 blocks(block_x, block_y);   // 2D block (32, 6)

    // compute grid dimensions
    const unsigned int grid_x = ceil(float(nx)/(float)blocks.x);
    const unsigned int grid_y = ceil(float(ny)/(float)blocks.y);
    dim3 grids(grid_x, grid_y);      // 2D grid

    event_pair timer;
    start_timer(&timer);

    for(int i = 0; i < params.iters(); ++i) {
        // update the values on the boundary only
        BC.updateBC(next_grid.dGrid_, curr_grid.dGrid_);

        // apply stencil
        switch (order) {
            case 2:
                gpuStencil<2><<<grids, blocks>>>(next_grid.dGrid_, curr_grid.dGrid_,
                                                 gx, nx, ny, xcfl, ycfl);
                break;
            case 4:
                gpuStencil<4><<<grids, blocks>>>(next_grid.dGrid_, curr_grid.dGrid_,
                                                 gx, nx, ny, xcfl, ycfl);
                break;
            case 8:
                gpuStencil<8><<<grids, blocks>>>(next_grid.dGrid_, curr_grid.dGrid_,
                                                 gx, nx, ny, xcfl, ycfl);
                break;
            default:
                fprintf(stderr, "%s\n", "Unknown order specified.");
        }

        check_launch("gpuStencil");

        Grid::swap(curr_grid, next_grid);
    }

    return stop_timer(&timer);
}


/**
 * Kernel to propagate finite difference grid from the current
 * time point to the next.
 *
 * This kernel should be optimized to compute finite difference updates
 * in blocks of size (blockDim.y * numYPerStep) * blockDim.x. Each thread
 * should calculate at most numYPerStep updates. It should still only use
 * global memory.
 *
 * @param next[out] Next grid state.
 * @param curr Current grid state.
 * @param gx Number of grid points in the x dimension.
 * @param nx Number of grid points in the x dimension to which the full
 *           stencil can be applied (ie the number of points that are at least
 *           order/2 grid points away from the boundary).
 * @param ny Number of grid points in the y dimension to which th full
 *           stencil can be applied.
 * @param xcfl Courant number for x dimension.
 * @param ycfl Courant number for y dimension.
 */
template<int order, int numYPerStep>
__global__
void gpuStencilLoop(float* next, const float* curr, int gx, int nx, int ny,
                    float xcfl, float ycfl) {
    // assert((gx - nx) == order);
    // thread id inside (nx * ny) area
    int tid_x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int tid_y = (blockIdx.y * blockDim.y) * numYPerStep + threadIdx.y;

    // multiple-pass update
    for (int i = 0; i < numYPerStep; ++i) {
        if (tid_x < nx && tid_y < ny) {
            // thread id inside (gx * gy) area (pad with border)
            // NOTE: do not pass template arguments directly
            int gid_x = tid_x + (gx - nx) / 2;
            int gid_y = tid_y + (gx - nx) / 2;
            int index = gid_x + gx * gid_y;
            next[index] = Stencil<order>(curr + index, gx, xcfl, ycfl);
            tid_y += blockDim.y;
        }
    }
}

/**
 * Propagates the finite difference 2D heat diffusion solver
 * using the gpuStencilLoop kernel.
 *
 * Use this function to do necessary setup and propagate params.iters()
 * number of times.
 *
 * @param curr_grid The current state of the grid.
 * @param params Parameters for the finite difference computation.
 * @returns Time required for computation.
 */
double gpuComputationLoop(Grid& curr_grid, const simParams& params) {

    boundary_conditions BC(params);

    Grid next_grid(curr_grid);

    // declare variables and compute parameters
    const int nx = params.nx(), ny = params.ny();
    const double xcfl = params.xcfl(), ycfl = params.ycfl();
    const int gx = params.gx();
    const int order = params.order();

    // choose block size as 256 (~192) threads (so that block_x is exact multiples of block_y)
    const int thread_num = 256;
    const int block_x = 32;
    const int block_y = thread_num / block_x;
    dim3 blocks(block_x, block_y);   // 2D block (32, 8)

    // compute stride in y direction of grid
    const int numYPerStep = block_x / block_y;

    // compute grid dimensions
    const unsigned int grid_x = ceil(float(nx)/(float)blocks.x);
    const unsigned int grid_y = ceil(float(ny)/(float)(blocks.y * numYPerStep));
    dim3 grids(grid_x, grid_y);      // 2D grid

    event_pair timer;
    start_timer(&timer);

    for(int i = 0; i < params.iters(); ++i) {

        // update the values on the boundary only
        BC.updateBC(next_grid.dGrid_, curr_grid.dGrid_);

        // apply stencil
        switch (order) {
            case 2:
                gpuStencilLoop<2, numYPerStep><<<grids, blocks>>>(next_grid.dGrid_, curr_grid.dGrid_,
                                                                  gx, nx, ny, xcfl, ycfl);
                break;
            case 4:
                gpuStencilLoop<4, numYPerStep><<<grids, blocks>>>(next_grid.dGrid_, curr_grid.dGrid_,
                                                                  gx, nx, ny, xcfl, ycfl);
                break;
            case 8:
                gpuStencilLoop<8, numYPerStep><<<grids, blocks>>>(next_grid.dGrid_, curr_grid.dGrid_,
                                                                  gx, nx, ny, xcfl, ycfl);
                break;
            default:
                fprintf(stderr, "%s\n", "Unknown order specified.");
        }

        check_launch("gpuStencilLoop");

        Grid::swap(curr_grid, next_grid);
    }

    return stop_timer(&timer);
}

/**
 * Kernel to propagate finite difference grid from the current
 * time point to the next.
 *
 * This kernel should be optimized to compute finite difference updates
 * in blocks of size side * side using shared memory.
 *
 * @param next[out] Next grid state.
 * @param curr Current grid state.
 * @param gx Number of grid points in the x dimension.
 * @param gy Number of grid points in the y dimension.
 * @param xcfl Courant number for x dimension.
 * @param ycfl Courant number for y dimension.
 */
template<int side, int order>
__global__
void gpuShared(float* next, const float* curr, int gx, int gy,
               float xcfl, float ycfl) {
    // TODO
}

/**
 * Propagates the finite difference 2D heat diffusion solver
 * using the gpuShared kernel.
 *
 * Use this function to do necessary setup and propagate params.iters()
 * number of times.
 *
 * @param curr_grid The current state of the grid.
 * @param params Parameters for the finite difference computation.
 * @returns Time required for computation.
 */
template<int order>
double gpuComputationShared(Grid& curr_grid, const simParams& params) {

    boundary_conditions BC(params);

    Grid next_grid(curr_grid);

    // TODO: Declare variables/Compute parameters.
    dim3 threads(0, 0);
    dim3 blocks(0, 0);

    event_pair timer;
    start_timer(&timer);

    for(int i = 0; i < params.iters(); ++i) {

        // update the values on the boundary only
        BC.updateBC(next_grid.dGrid_, curr_grid.dGrid_);

        // TODO: Apply stencil.

        check_launch("gpuShared");

        Grid::swap(curr_grid, next_grid);
    }

    return stop_timer(&timer);
}

